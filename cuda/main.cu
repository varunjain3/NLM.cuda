#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include "helper.h"

#define imgH 1024
#define imgW 1024
#define sWindowSize 7
#define nWindowSize 3
#define image_path "../sp_noise/Image3.png"

using namespace std;

// #def data folder

// run using: g++ -std=c++11 main.cpp -o main `pkg-config --cflags --libs opencv`
// run using: nvcc -std=c++11 main.cpp -o main `pkg-config --cflags --libs opencv`

// /*
__global__ void pixel_kernel_call(float* paddedImage, float* outputImage, int rows, int cols, int windowSize, int searchWindowSize, int h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int halfWindowSize = windowSize / 2;
    int halfSearchWindowSize = searchWindowSize / 2;

    float weightedSum = 0;
    float similaritySum = 0;

    for(int k=-halfSearchWindowSize; k<=halfSearchWindowSize; k++){
        for(int l=-halfSearchWindowSize; l<=halfSearchWindowSize; l++){
            float dist = 0;
            for(int m=-halfWindowSize; m<=halfWindowSize; m++){
                for(int n=-halfWindowSize; n<=halfWindowSize; n++){
                    dist += pow(paddedImage[(i+k+halfSearchWindowSize)*cols + (j+l+halfSearchWindowSize)] - paddedImage[(i+m+halfSearchWindowSize)*cols + (j+n+halfSearchWindowSize)], 2);
                }
            }
            // cout<<dist<<endl;
            dist = sqrt(dist);
            // cout<<dist<<endl;
            float w = exp(-dist/(h));

            weightedSum += w*paddedImage[(i+k+halfSearchWindowSize)*cols + (j+l+halfSearchWindowSize)];
            similaritySum += w;
        }
    }
    float intensity = weightedSum/similaritySum;
    // cout<<intensity<<endl;
    outputImage[i*cols + j] = intensity;
}

// */

cv::Mat NL_Means(cv::Mat src, int h = 2, int windowSize = 3, int searchWindowSize = 7)
{
    int rows = src.rows;
    int cols = src.cols;

    int halfWindowSize = windowSize / 2;
    int halfSearchWindowSize = searchWindowSize / 2;

    cout << "Performing NL_Means on the Image" << endl;

    vector<vector<float>> paddedImage = padImage(src, searchWindowSize);

    paddedImage = floatImage(paddedImage);

    float *outputImage;
    float *dev_pad, *dev_out_img;


    // vector<vector<float>> outputImage

    // size_t N = 128;
    size_t pad_arr_len = paddedImage.size() * paddedImage[0].size();
    
    //create buffer on device
    hipError_t err = hipMalloc(&dev_pad, pad_arr_len*sizeof(float));
    if (err != hipSuccess){
        cout<<"Dev Memory not allocated"<<endl;
        exit(-1);
    }
    hipError_t err = hipMalloc(&dev_out_img, pad_arr_len*sizeof(float));
    if (err != hipSuccess){
        cout<<"Dev Memory not allocated"<<endl;
        exit(-1);
    }

    hipMemcpy(dev_pad, paddedImage, pad_arr_len * sizeof(float), hipMemcpyHostToDevice);
    size_t threads = 16;
    dim3 threadsperblock(threads, threads);
    dim3 numBlocks(threads*threads/threadsperblock.x,  threads*threads/threadsperblock.y);

    // print parameters
    cout << "Threads: " << threads << endl;
    cout << "Blocks: " << numBlocks.x << "x" << numBlocks.y << endl;
    cout << "Threads per block: " << threadsperblock.x << "x" << threadsperblock.y << endl;

    for (int i = 0; i < rows; i++)
    {
        cout << i << endl;
        for (int j = 0; j < cols; j++)
        {
            pixel_kernel_call<<<blockspergrid, threadsperblock>>>(dev_pad, dev_out_img, cols, windowSize, searchWindowSize, h);
        }
    }

    hipMemcpy(outputImage, dev_out_img, pad_arr_len * sizeof(float), hipMemcpyHostToDevice);

    cout << "Done" << endl;

    // outputImage = intImage(outputImage);

    // cv::Mat dst = Vec2Mat(outputImage, "outputImage.png");

    return src;
}

int main(int argc, char **argv)
{
    int searchWindowSize = sWindowSize;
    int windowSize = nWindowSize;

    // string image_path
    cout << "Loading image " << image_path << endl;

    cv::Mat src = cv::imread(image_path, cv::IMREAD_GRAYSCALE);
    // save original shape of image
    int OrignalH = src.rows;
    int OrignalW = src.cols;

    cv::resize(src, src, cv::Size(imgH - 2 * searchWindowSize, imgW - 2 * searchWindowSize));


    

    cout << "Shape of image: " << src.size() << endl;

    if (src.empty())
    {
        cout << "Could not open or find the image!\n"
             << endl;
        cout << "Usage: " << argv[0] << " <Input image>" << endl;
        return -1;
    }
    
    cv::Mat dst = NL_Means(src, 2, windowSize, searchWindowSize);

    cv::resize(dst, dst, cv::Size(OrignalH, OrignalW));
    cv::imwrite("output.png", dst);

    return 0;
}